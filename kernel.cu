#include "hip/hip_runtime.h"
#include ""
#include "gputimer.h"

#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>

#include "A_10.h";
#include "b_10.h";

#define N 10

using namespace std;

constexpr auto MAX_NUMBER_THREADS = 1024;

hipError_t solveMatrixWithCuda(float* matrixA, float* vectorB, int dimension, int numOfThreads);

//__global void solveMatrixKernel(float** inverseMatrix, float** vector, int dimension, int numOfThreads) 
//{
//	
//
//}

// Function to get cofactor of A[p][q] in temp[][]. n is current 
// dimension of A[][] 
void getCofactor(float matrix[N][N], float temp[N][N], int p, int q, int n)
{
	int i = 0, j = 0;

	// Looping for each element of the matrix 
	for (int row = 0; row < n; row++)
	{
		for (int col = 0; col < n; col++)
		{
			//  Copying into temporary matrix only those element 
			//  which are not in given row and column 
			if (row != p && col != q)
			{
				temp[i][j++] = matrix[row][col];

				// Row is filled, so increase row index and 
				// reset col index 
				if (j == n - 1)
				{
					j = 0;
					i++;
				}
			}
		}
	}
}

/* Recursive function for finding determinant of matrix.
   n is current dimension of A[][]. */
int determinant(float matrix[N][N], int n)
{
	int D = 0; // Initialize result 

	//  Base case : if matrix contains single element 
	if (n == 1)
		return matrix[0][0];

	float temp[N][N]; // To store cofactors 

	int sign = 1;  // To store sign multiplier 

	 // Iterate for each element of first row 
	for (int f = 0; f < n; f++)
	{
		// Getting Cofactor of A[0][f] 
		getCofactor(matrix, temp, 0, f, n);
		D += sign * matrix[0][f] * determinant(temp, n - 1);

		// terms are to be added with alternate sign 
		sign = -sign;
	}

	return D;
}

// Function to get adjoint of A[N][N] in adj[N][N]. 
void adjoint(float matrix[N][N], float adj[N][N])
{
	if (N == 1)
	{
		adj[0][0] = 1;
		return;
	}

	// temp is used to store cofactors of A[][] 
	int sign = 1; 
	float temp[N][N];

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			// Get cofactor of A[i][j] 
			getCofactor(matrix, temp, i, j, N);

			// sign of adj[j][i] positive if sum of row 
			// and column indexes is even. 
			sign = ((i + j) % 2 == 0) ? 1 : -1;

			// Interchanging rows and columns to get the transpose of the cofactor matrix 
			adj[j][i] = (sign) * (determinant(temp, N - 1));
		}
	}
}

// Function to calculate and store inverse, returns false if 
// matrix is singular 
bool inverse(float matrix[N][N], float inverse[N][N])
{
	// Find determinant of A[][] 
	int det = determinant(matrix, N);
	if (det == 0)
	{
		cout << "Singular matrix, can't find its inverse";
		return false;
	}

	// Find adjoint 
	float adj[N][N];
	adjoint(matrix, adj);

	// Find Inverse using formula "inverse(A) = adj(A)/det(A)" 
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			inverse[i][j] = adj[i][j] / float(det);
		}
	}

	return true;
}
 
void display(float matrix[N][N])
{
	for (int i = 0; i < N; i++) {
		cout << "{ ";
		for (int j = 0; j < N; j++) {
			cout << matrix[i][j] << ", ";
		}
		cout << " }" << endl;
	}
}


int main(int argc, char* argv[])
{
	int numOfThreads = 10;
	int dimension = 10;

	float adj[N][N] = { 0 };  // To store adjoint of A[][] 

	float inv[N][N] = { 0 }; // To store inverse of A[][] 
	
	//hipError_t status = adjointWithCuda(numOfThreads, &InputMatrix);

	cout << "\nDimension of A = " << N;
	cout << "\nNumber of Threads = " << numOfThreads;

	cout << "\nMatrix A =\n";
	display(A_10);

	cout << "\nThe Adjoint of A=\n";
	adjoint(A_10, adj);
	display(adj);

	cout << "\nThe Inverse of A=\n";
	if (inverse(A_10, inv))
		display(inv);

	cout << "\nx = Inverse of A * b = ";

	return 0;

	//if (argc != 5 || argv[1] == NULL || argv[2] == NULL || argv[3] == NULL || argv[4] == NULL ||
	//	argv[1] == "-h" || argv[1] == "--help" || argv[1] == "--h") {
	//	cout << "ParallelMatrixSolver.exe <Matrix A> <Vector B> <# threads>" << endl;
	//	return 0;
	//}
	//else {
	//	if (argv[2] != NULL) {
	//		inputImgName = argv[2];
	//	}
	//	if (argv[3] != NULL) {
	//		outImgName = argv[3];
	//	}
	//	if (argv[4] != NULL) {
	//		numOfThreads = stoi(argv[4]);
	//	}
	//}

	//if (argv[1] != NULL && !strcmp(argv[1], "pool")) {
	//	cout << "Pooling" << endl;
	//	hipError_t status = imagePoolingWithCuda(numOfThreads, inputImgName, outImgName);
	//}

	//return 0;
}

hipError_t solveMatrixWithCuda(float* inverseA, float* vectorB, int dimension, int numOfThreads) {
	hipError_t cudaStatus = hipError_t::cudaErrorDeviceUninitilialized;
	GpuTimer gpuTimer; // Struct for timing the GPU

	float* dev_inverseA;
	float* dev_vectorB;
	float* dev_solution;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate memory for the input matrix, then it's adjoint, then it's inverse
	cudaStatus = hipMallocManaged((void**)& dev_inverseA, dimension * dimension * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMallocManaged((void**)& dev_vectorB, dimension * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMallocManaged((void**)& dev_solution, dimension * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy over values from the input matrices to the CUDA memory
	// TODO

	int numBlocks = ((numOfThreads + (MAX_NUMBER_THREADS - 1)) / MAX_NUMBER_THREADS);
	int threadsPerBlock = ((numOfThreads + (numBlocks - 1)) / numBlocks);

	/*************************************** Parrallel Part of Execution **********************************************/
	gpuTimer.Start();
	//solveMatrixKernel << <numBlocks, threadsPerBlock >> > (dev_inverseA, dev_vectorB, dev_solution, dimension, threadsPerBlock);
	gpuTimer.Stop();
	/******************************************************************************************************************/
	printf("-- Number of Threads: %d -- Execution Time (ms): %g \n", numOfThreads, gpuTimer.Elapsed());
	
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "solveMatrixWithCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching one of the kernels!\n", cudaStatus);
		goto Error;
	}

Error:
	hipFree(dev_inverseA);
	hipFree(dev_vectorB);
	hipFree(dev_solution);
	return cudaStatus;
}
